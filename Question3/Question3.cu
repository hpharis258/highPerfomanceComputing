/*************************************************************
 * Question 3: Password Cracking using CUDA.                                         
 * By: Haroldas Varanauskas
 * Student Number: 2411253
 *************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Cuda Crypt Function
__device__
char* CudaCrypt(char* rawPassword) {
	
	// Allocate memory for password
	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking numbers
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

// Function to check if two strings are the same.
__device__ 
int isStringMatching(char* StringOne, char* StringTwo, int length) {
	int result = 1;
	for (int i = 0; i < length; i++) {
		if (StringOne[i] != StringTwo[i]) {
			result = 0;
			break;
		}
	}
	return result;
}

// DECRYPT PASSWORD FUNCTION
__global__
void decryptPassword(char* alphabet, char* numbers, char* encryptedPass, char* deviceOutputPass)
{
	/// Get cuda unique thread id
	int uniqueid = blockDim.x * blockIdx.x + threadIdx.x;
	
	/// Check if another thread found output pass before starting
	if(*deviceOutputPass != NULL) {
		// Early Return Another Thread Found Password.
		return;
	}

	/// Create pass to check on this thread
	char GeneratedPass[4];
	GeneratedPass[0] = alphabet[blockIdx.x];
	GeneratedPass[1] = alphabet[blockIdx.y];
	GeneratedPass[2] = numbers[threadIdx.x];
	GeneratedPass[3] = numbers[threadIdx.y];
	
	/// Encrypt the potential password
	char* encryptedPotentialPassword;
	encryptedPotentialPassword = CudaCrypt(GeneratedPass);
	
	/// check if encrypted password string is the same as the encrypted potential password string.
	if (isStringMatching(encryptedPass, encryptedPotentialPassword, 11) > 0 )
	{
		/// Password strings are the same so set deviceOutputPassword to the current generated password
		printf("THREAD ID '%d' Encrypted password '%s' : '%s' matches password = '%s'\n", uniqueid, encryptedPass, GeneratedPass, encryptedPotentialPassword);
		// Loop to copy the string
		for (int i = 0; i < 4; i++ ) {
			deviceOutputPass[i] = GeneratedPass[i];
		}
	}
}

/**
	
*/
int main(int argc, char** argv) {
	printf("Welcome to the CUDA password cracker!!!\n"); 
	//Test Encrypted Passwords:
	//aa52 = ccbddb7362
	char* encryptedPass;
	// pass in the encrypted password as a command line argument if you want to. 
	if (argc > 1) {
		encryptedPass = argv[1];
	}else{
	 encryptedPass = "ccbddb7362";
	}
	 
	printf("Encrypted Password used: '%s'\n", encryptedPass);
	
	// Init abc and numbers array to read only use in cuda 
	char cpuABC[26] = { 'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z' };
	char cpuNumbers[10] = { '0', '1', '2', '3', '4', '5', '6' ,'7', '8', '9' };
	
	int sizeOfEncryptedPass = sizeof(char) * 11;
	
	char* gpuABC;
	hipMalloc( (void**) &gpuABC, sizeof(char) * 26 );
	hipMemcpy( gpuABC, cpuABC, sizeof(char) * 26, hipMemcpyHostToDevice );
	
	char* gpuNumbers;
	hipMalloc( (void**) &gpuNumbers, sizeof(char) * 10 );
	hipMemcpy( gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice );
	
	char* gpuEncryptedPass;
	hipMalloc( (void**) &gpuEncryptedPass, sizeOfEncryptedPass );
	hipMemcpy( gpuEncryptedPass, encryptedPass, sizeOfEncryptedPass, hipMemcpyHostToDevice);
	
	char* gpuOutputPass;
	hipMalloc( (void**) &gpuOutputPass, sizeOfEncryptedPass );
	
	// 26 block for x, 26 block for y, Blocks divide the letters, threads divide the numbers
	decryptPassword<<< dim3(26, 26, 1), dim3(10, 10, 1) >>>(gpuABC, gpuNumbers, gpuEncryptedPass, gpuOutputPass);
	// Synchronize threads
	hipDeviceSynchronize();
	// Allocate Memory for the Output password.
	char* cpuOutputPass = (char*)malloc( sizeof(char) * 4 );
	/// Copy GPU output pass to the CPU : Device to Host
	hipMemcpy(cpuOutputPass, gpuOutputPass, sizeOfEncryptedPass, hipMemcpyDeviceToHost);
	/// If output password is not null or empty, print output password
	if (cpuOutputPass != NULL && cpuOutputPass[0] != 0) {
		printf("Given Encrypted Password: '%s'\n", encryptedPass);
		printf("Found Decrypted Password: '%s'\n", cpuOutputPass);
	} else {
		printf("Can't find a password.\n");
	}
	/// Free all of the used Memory
	hipFree(gpuABC);
	hipFree(gpuNumbers);
	hipFree(gpuEncryptedPass);
	hipFree(gpuOutputPass);
	free(cpuOutputPass);
	return 0;
}