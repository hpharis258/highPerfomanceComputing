/*************************************************************
 * Question 3: Password Cracking using CUDA.                                         
 * By: Haroldas Varanauskas
 * Student Number: 2411253
 *************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Encrypt Password function
__device__
char* CudaCrypt(char* rawPassword) {
	
	// Allocate memory for password
	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking numbers
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

/**
	Checks if one char string matches another char string of the length
*/
__device__ 
int isEncryptedMatching(char* one, char* two, int length) {
	int result = 1;
	for (int i = 0; i < length; i++) {
		if (one[i] != two[i]) {
			result = 0;
			break;
		}
	}
	return result;
}

/**
	Decrypts a pass using a CUDA thread
*/
__global__
void decryptPass(char* alphabet, char* numbers, char* encryptedPass, char* deviceOutputPass)
{
	/// Get cuda unique thread id
	int uid = blockDim.x * blockIdx.x + threadIdx.x;
	
	/// Check if another thread found output pass before starting
	if(*deviceOutputPass != NULL) {
		// Early Return Another Thread Found Password.
		return;
	}

	/// Create password to check on this thread
	char potentialPass[4];
	potentialPass[0] = alphabet[blockIdx.x];
	potentialPass[1] = alphabet[blockIdx.y];
	potentialPass[2] = numbers[threadIdx.x];
	potentialPass[3] = numbers[threadIdx.y];
	
	/// Encrypt the potential password
	char* encryptedPotential;
	encryptedPotential = CudaCrypt(potentialPass);
	
	//printf("UID: '%d' Plain: '%s' Encrypted Plain: '%s' Target Encrypted: '%s'\n", uid, potentialPass, encryptedPotential, encryptedPass);
	
	/// Check the current potential pass is matches the target encryptedPass
	if ( isEncryptedMatching(encryptedPass, encryptedPotential, 11) > 0 )
	{
		/// Matches so set deviceOutputPassword to the current combination
		printf("UID '%d' Encrypted pass '%s' from combination '%s' matches pass = '%s'\n", uid, encryptedPass, potentialPass, encryptedPotential);
		for (int i = 0; i < 4; i++ ) {
			deviceOutputPass[i] = potentialPass[i];
		}
	}
}

/**
	
*/
int main(int argc, char** argv) {
	// TEST Password used: 
	/*
		Test Encrypted Passwords:
		az01 = ccbdwy2253
		aa52 = ccbddb7362
	*/
	// HARD CODED TEST PASS
	char* encryptedPass = "ccbdwy2253";
	// optionaly you can pass in the 
	if (argc > 1) {
		encryptedPass = argv[1];
	}
	 
	printf("Pass: '%s'\n", encryptedPass);
	
	// Init alphabet and numbers array to read only use in cuda 
	char cpuAlphabet[26] = { 'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z' };
	char cpuNumbers[10] = { '0', '1', '2', '3', '4', '5', '6' ,'7', '8', '9' };
	
	int sizeOfEncryptedPass = sizeof(char) * 11;
	
	char* gpuAlphabet;
	hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26 );
	hipMemcpy( gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice );
	
	char* gpuNumbers;
	hipMalloc( (void**) &gpuNumbers, sizeof(char) * 10 );
	hipMemcpy( gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice );
	
	char* gpuEncryptedPass;
	hipMalloc( (void**) &gpuEncryptedPass, sizeOfEncryptedPass );
	hipMemcpy( gpuEncryptedPass, encryptedPass, sizeOfEncryptedPass, hipMemcpyHostToDevice);
	
	char* gpuOutputPass;
	hipMalloc( (void**) &gpuOutputPass, sizeOfEncryptedPass );
	
	
	/// Launch cuda threads and await finish
	decryptPass<<< dim3(26, 26, 1), dim3(10, 10, 1) >>>(gpuAlphabet, gpuNumbers, gpuEncryptedPass, gpuOutputPass);
	hipDeviceSynchronize();
	
	
	printf("Finished synchronizing CUDA threads\n");
	
	/// Copy GPU output pass to the CPU
	char* cpuOutputPass = (char*)malloc( sizeof(char) * 4 );
	hipMemcpy(cpuOutputPass, gpuOutputPass, sizeOfEncryptedPass, hipMemcpyDeviceToHost);

	/// If output pass contained an output, print the results
	printf("---\n");
	printf("Results:\n");
	if (cpuOutputPass != NULL && cpuOutputPass[0] != 0) {
		printf("Given Encrypted Pass: '%s'\n", encryptedPass);
		printf("Found Decrypted Pass: '%s'\n", cpuOutputPass);
	} else {
		printf("Unable to determine a password.\n");
	}
	
	/// Free any malloc'd memory
	hipFree(gpuAlphabet);
	hipFree(gpuNumbers);
	hipFree(gpuEncryptedPass);
	hipFree(gpuOutputPass);
	free(cpuOutputPass);
}